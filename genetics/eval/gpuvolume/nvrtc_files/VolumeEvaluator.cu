#include "hip/hip_runtime.h"
/*
 * VolumeEvaluator.cu
 *
 *  Created on: Dec 20, 2015
 *      Author: Karl Haubenwallner
 */

#include "combined_header.cuh"

#define EPSILON 0.0000001f

struct Object {
	unsigned int triangle_offset;
	unsigned int num_triangles;
	unsigned int matrix_offset;
};

struct Triangle {
	math::float3 v0;
	math::float3 v1;
	math::float3 v2;
};


__device__ bool ray_triangle_intersect(const math::float3& O, const math::float3& D, const math::float3& v0, const math::float3& v1, const math::float3& v2)
{

	// Moeller / Trumbore ray-triangle intersection algorithm

	math::float3 E1 = v1 - v0;
	math::float3 E2 = v2 - v0;

	// q = P
	math::float3 P = cross(D, E2);

	// a = det
	float det = dot(E1, P);

	if ((det > -EPSILON && det < EPSILON)) {
		return false;;
	}

	math::float3 T = (O - v0);
	math::float3 Q = cross(T, E1);

	math::float3 result(dot(E2, Q), dot(T, P), dot(D, Q));
	result = result * 1.0f/det;

	bool miss = false;

	miss = (result.y < 0.0f || result.y > 1.0f);

	miss |= (result.z < 0.0f || (result.y + result.z) > 1.0f);

	miss |= (result.x < 0.0f);

	return !miss;
}

#ifdef not_used
__global__ void d_calc_voxels_raystabbing(	const Triangle* triangles,
		const unsigned int num_triangles,
		unsigned int* voxel_outline,
		unsigned int* voxel_volume,
		math::VoxelCoords *vc)
{

	math::int3 voxel;
	voxel.x = threadIdx.x + blockIdx.x * blockDim.x;
	voxel.y = threadIdx.y + blockIdx.y * blockDim.y;
	voxel.z = 0;

	// find all triangles that intersect with this voxel line

	math::float3 origin = vc->toWorld(voxel);
	math::float3 direction(0.0f, 0.0f, 1.0f);

	__syncthreads();
	for (int i = 0; i < num_triangles; ++i) {
		math::float3 intersection;

		if (ray_triangle_intersect(origin, direction, triangles[i].v0, triangles[i].v1, triangles[i].v2, intersection)) {
			int index = vc->toIndex(vc->toVoxel(intersection));
			voxel_outline[index] = 1;
		}
	}

	__syncthreads();


	// do first flood-fill

	unsigned short val = 0;
	for (voxel.z = 0; voxel.z < vc->voxelPerDimension().z; voxel.z++) {

		int index = vc->toIndex(voxel);

		val += voxel_outline[index];

		voxel_volume[index] = clamp(val, 0, 1);

		val = val % 2;
	}

	// second flood fill from the other direction, to prevent single-voxel-triangles from flood-filling to the end
	val = 0;
	for (voxel.z = vc->voxelPerDimension().z -1; voxel.z >= 0; voxel.z--) {
		int index = vc->toIndex(voxel);

		val += voxel_outline[index];

		voxel_volume[index] += clamp(val, 0, 1);

		val = val % 2;
	}
}

#endif

__global__ void d_calc_voxels_objects(
		const Object* objects,
		int num_objects,
		Triangle* triangles,
		math::float3x4* matrices,
		unsigned int* voxel_volume,
		int *outside_voxels,
		math::VoxelCoords *vc)
{

	math::VoxelCoords local_vc = *vc;
	math::float3x4 instance = matrices[blockIdx.x];
	Object object = objects[0];
	for (int i = 0; i < num_objects; ++i) {
		if (objects[i].triangle_offset < blockIdx.x) {
			object = objects[i];
		}
	}
	__shared__ math::int3 s_min;
	__shared__ math::int3 s_max;

	Triangle triangle = triangles[object.triangle_offset];

	// apply the matrix to the triangle
	triangle.v0 = instance * math::float4(triangle.v0, 1.0f);
	triangle.v1 = instance * math::float4(triangle.v1, 1.0f);
	triangle.v2 = instance * math::float4(triangle.v2, 1.0f);

	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		s_min = local_vc.toVoxel(triangle.v0);
		s_max = local_vc.toVoxel(triangle.v0);
	}

	__syncthreads();


	// calc the bounding box of the object
	if (threadIdx.y == 0) {

		for (int offset = threadIdx.x; offset < object.num_triangles; offset +=  blockDim.x) {
			int triangle_index = object.triangle_offset + offset;
			Triangle t = triangles[triangle_index];

			// apply the matrix to the triangle
			t.v0 = instance * math::float4(t.v0, 1.0f);
			t.v1 = instance * math::float4(t.v1, 1.0f);
			t.v2 = instance * math::float4(t.v2, 1.0f);

			math::int3 v0 = local_vc.toVoxel(t.v0);
			math::int3 v1 = local_vc.toVoxel(t.v1);
			math::int3 v2 = local_vc.toVoxel(t.v2);

			atomicMin(&s_min.x, min(v0.x, min(v1.x, v2.x)));
			atomicMin(&s_min.y, min(v0.y, min(v1.y, v2.y)));
			atomicMin(&s_min.z, min(v0.z, min(v1.z, v2.z)));

			atomicMax(&s_max.x, max(v0.x, max(v1.x, v2.x)));
			atomicMax(&s_max.y, max(v0.y, max(v1.y, v2.y)));
			atomicMax(&s_max.z, max(v0.z, max(v1.z, v2.z)));

		}
	}
	__syncthreads();

	// find out how many voxels we have to check
	math::int3 span = (s_max - s_min) + math::int3(1, 1, 1);
	int num_voxels = local_vc.calcNumVoxels(span);
	math::float3 ray_direction(0.0f, 1.0f, 0.0f);
	
	//if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
	//	printf("span(%d, %d, %d), num voxels = %d, s_min(%d, %d, %d)\n", span.x, span.y, span.z, num_voxels, s_min.x, s_min.y, s_min.z);
	//}
	

	for (int index = threadIdx.x; index < num_voxels; index += blockDim.x) {
		math::int3 voxel_coord = local_vc.getVoxelCoordsInsideBox(index, span) + s_min;
		if (local_vc.check(voxel_coord) == false) {
			atomicAdd(outside_voxels, 1);
		}
		else {
			//int voxel_index = local_vc.toIndex(voxel_coord);
			math::float3 voxel_center = local_vc.toWorld(voxel_coord);
			bool hits = false;
			for (int i = 0; i < object.num_triangles; ++i) {
				Triangle t = triangles[object.triangle_offset + i];
				// apply the matrix to the triangle
				t.v0 = instance * math::float4(t.v0, 1.0f);
				t.v1 = instance * math::float4(t.v1, 1.0f);
				t.v2 = instance * math::float4(t.v2, 1.0f);
				// count triangle check if the voxel is inside the object
				if (ray_triangle_intersect(voxel_center, ray_direction, t.v0, t.v1, t.v2) == true) {
					hits  = !hits;
				}
			}
			if (hits) {
				//printf("%d %d %d -> %d\n", voxel_coord.x, voxel_coord.y, voxel_coord.z, local_vc.toIndex(voxel_coord));
				atomicAdd(&voxel_volume[local_vc.toIndex(voxel_coord)], 1);
			}
		}
	}

//	__syncthreads();

}


__global__ void d_count_voxels(unsigned int* voxels, unsigned int* num_voxel, math::VoxelCoords* vc)
{
	math::int3 voxel;
	voxel.x = threadIdx.x + blockIdx.x * blockDim.x;
	voxel.y = threadIdx.y + blockIdx.y * blockDim.y;
	voxel.z = threadIdx.z + blockIdx.z * blockDim.z;

	__shared__ int num;

	int index = vc->toIndex(voxel);
	if (voxels[index] > 0) {
		atomicAdd(&num, 1);
	}
	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		atomicAdd(num_voxel, num);
	}
}


__global__ void d_compare_voxels(unsigned int* voxels_target, unsigned int* voxels_object, int* ratings, math::VoxelCoords* vc)
{
	math::int3 voxel;
	voxel.x = threadIdx.x + blockIdx.x * blockDim.x;
	voxel.y = threadIdx.y + blockIdx.y * blockDim.y;
	voxel.z = threadIdx.z + blockIdx.z * blockDim.z;

	__shared__ int good;
	__shared__ int bad;
	__shared__ int overlap;

	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		good = 0;
		bad = 0;
		overlap = 0;
	}

	__syncthreads();

	int index = vc->toIndex(voxel);

	bool v_t = (voxels_target[index] != 0);
	bool v_o = (voxels_object[index] != 0);
	bool v_overlap = (voxels_object[index] > 1);

	voxels_object[index] = 0;

	if (v_t && v_o) {
		atomicAdd(&good, 1);
	}
	else if (v_o && !v_t) {
		atomicAdd(&bad, 1);
	}
	if (v_overlap) {
		atomicAdd(&overlap, 1);
	}

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		atomicAdd(&ratings[0], good);
		atomicAdd(&ratings[1], bad);
		atomicAdd(&ratings[2], overlap);
	}

}

__global__ void d_export_voxels(unsigned int* voxel_space, math::float4* voxels, unsigned int* num_voxels, math::VoxelCoords* vc)
{
	math::int3 voxel;
	voxel.x = threadIdx.x + blockIdx.x * blockDim.x;
	voxel.y = threadIdx.y + blockIdx.y * blockDim.y;
	voxel.z = threadIdx.z + blockIdx.z * blockDim.z;

	unsigned int index = vc->toIndex(voxel);
	unsigned int voxel_val = voxel_space[index];
	if (voxel_val > 0) {
		unsigned int offset = atomicAdd(num_voxels, 1);
		math::float3 world_coords = vc->toWorld(voxel);
		math::float4 voxel_data;
		voxel_data.x = world_coords.x;
		voxel_data.y = world_coords.y;
		voxel_data.z = world_coords.z;
		voxel_data.w = voxel_val * 1.0f;
		voxels[offset] = voxel_data;
	}
}

extern "C" {


__global__  void _fct_count_voxels(unsigned int* voxels, unsigned int* num_voxel, math::VoxelCoords *vc) {
	d_count_voxels(voxels, num_voxel, vc);
}

__global__ void _fct_compare_voxels(unsigned int* voxels_target, unsigned int* voxels_object, int* ratings, math::VoxelCoords *vc) {
	d_compare_voxels(voxels_target, voxels_object, ratings, vc);
}

__global__ void _fct_export_voxels(unsigned int* voxels, math::float4* voxel_centers, unsigned int* num_voxels, math::VoxelCoords *vc) {
	d_export_voxels(voxels, voxel_centers, num_voxels, vc);
}

__global__ void _fct_calc_voxels(
		const Object* objects,
		int num_objects,
		Triangle* triangles,
		math::float3x4* matrices,
		unsigned int* voxel_volume,
		int *outside_voxels,
		math::VoxelCoords* vc)
{
	d_calc_voxels_objects(objects, num_objects, triangles, matrices, voxel_volume, outside_voxels, vc);
}

}

